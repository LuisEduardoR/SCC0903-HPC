
/*
  Universidade de Sao Paulo - ICMC
  SSC0903 - Computacao de Alto Desempenho
  Atividade Aula
  Grupo:
  João Pedro A. S. Secundino (10692054);
  João Pedro Uchôa Cavalcante (10801169);
  Luís Eduardo Rozante de Freitas Pereira (10734794);
  Sérgio Ricardo G. B. Filho (10408386);
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void sum_prod_scalar(int *a, int *b, int dim,int *result){
    __shared__ int row_sum;
    row_sum = 0;
    
    __syncthreads();

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    atomicAdd(&row_sum, a[id] * b[id]);
    
    __syncthreads();

    if(id == blockDim.x * blockIdx.x){
        atomicAdd(result, row_sum);
    }

}

// <<LENGTH, WIDTH>>
__global__ void find_min(int *A, int *B, int dim, int *result){
    __shared__ int row_min;
    row_min = 1000000;
    __syncthreads();

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < dim){
        if(A[id] < row_min)
            row_min = A[id];
        if(B[id] < row_min)
            row_min = A[id];
    }  

    __syncthreads();

    // se for primeira thread do bloco
    if(id == blockDim.x * blockIdx.x){
        if(row_min < *result) 
            *result = row_min; 
    }


}   

// <<LENGTH, WIDTH>>
__global__ void find_max(int *A, int *B, int dim, int *result){
    __shared__ int row_max;
    row_max = -1000000;
    
    __syncthreads();

    int id = blockIdx.x * blockDim.x + threadIdx.x;
 
    if(id < dim){
        if(A[id] > row_max)
            row_max = A[id];
        if(B[id] > row_max)
            row_max = A[id];
    }  

    __syncthreads();

    // se for primeira thread do bloco
    if(id == blockDim.x * blockIdx.x){
        if(row_max < *result) 
            *result = row_max; 
    }
}

int main(int argc, char **argv) {

    // Declara a dimensão da matriz
    int dim;

    // Declara as matrizes
    int *A,*B, *A_device_s1, *B_device_s1;

    // Declara o acumulador para o produto escalar global o maximo e o minimo
    int somape, minimo = 10000, maximo = -10000;

    // Declara um vetor para os produtos escalares locais
    int *prod_escalar;
    
    // Lê a dimensão das matrizes
    scanf(" %d", &dim);
    
    // Cria as streams
    hipStream_t stream1_soma, stream2_min, stream3_max;  
    hipStreamCreate(&stream1_soma);    // Stream que realiza a soma
    hipStreamCreate(&stream2_min);     // Stream que realiza o minímo
    hipStreamCreate(&stream3_max);     // Stream que realiza o máximo

    // Número de elementos da matriz.
    int tam = dim * dim;

    // Aloca as matrizes no host 
    hipHostMalloc((void**)&A, tam * (sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&B, tam * (sizeof(int)), hipHostMallocDefault);

    // Aloca as matrizes no device para cada stream
    hipMalloc((void**)&A_device_s1, tam * (sizeof(int))); 
    hipMalloc((void**)&B_device_s1, tam * (sizeof(int)));

    // Aloca um vetor para armazenar os produtos escalares de cada linha
    hipMalloc((void**)&prod_escalar, dim * (sizeof(int)));

    // Lê a matriz A
    for(int i = 0; i < dim; i++) {
        for(int j = 0; j < dim; j++) {
            scanf(" %d",&(A[i*dim+j]));
        }
    }

    // Lê a matriz B
    for(int i = 0; i < dim; i++) {
        for(int j = 0; j < dim; j++) {
            scanf(" %d",&(B[i*dim+j])); 
        }
    }
    
    // Copia a memória para cada stream do device
    hipMemcpyAsync(A_device_s1, A, tam * (sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpyAsync(B_device_s1, B, tam * (sizeof(int)), hipMemcpyHostToDevice);

    sum_prod_scalar <<<dim, dim, 0, stream1_soma>>> (A_device_s1, B_device_s1, tam, &somape);
    find_min <<<dim, dim, 0, stream2_min>>> (A_device_s1, B_device_s1, tam, &minimo);
    find_max <<<dim, dim, 0, stream3_max>>> (A_device_s1, B_device_s1, tam, &maximo);
    
    // Sincroniza as streams
    hipStreamSynchronize(stream1_soma);
    hipStreamSynchronize(stream2_min);
    hipStreamSynchronize(stream3_max);

    // Imprime o resultado
    printf("%d %d %d\n",somape, minimo, maximo); 
    
    // Desaloca as matrizes
    hipHostFree(A);
    hipHostFree(B);
    hipFree(A_device_s1);
    hipFree(B_device_s1);
    hipFree(prod_escalar);
    
    // Destroi as streams
    hipStreamDestroy(stream1_soma);
    hipStreamDestroy(stream2_min);
    hipStreamDestroy(stream3_max);

}
