
/*
  Universidade de Sao Paulo - ICMC
  SSC0903 - Computacao de Alto Desempenho
  Atividade Aula
  Grupo:
  João Pedro A. S. Secundino (10692054);
  João Pedro Uchôa Cavalcante (10801169);
  Luís Eduardo Rozante de Freitas Pereira (10734794);
  Sérgio Ricardo G. B. Filho (10408386);
*/

#include <stdlib.h>
#include <stdio.h>
#include <string.h>
#include <hip/hip_runtime.h>

__global__ void sum_prod_scalar(int *A, int *B, int dim, int *result){

    __shared__ int row_sum;
    row_sum = *result;
    
    __syncthreads();

    int id = blockDim.x * blockIdx.x + threadIdx.x;
    
    atomicAdd(&row_sum, A[id] * B[id]);
    
    __syncthreads();

    if(id == blockDim.x * blockIdx.x){
        atomicAdd(result, row_sum);
    }

}

// <<LENGTH, WIDTH>>
__global__ void find_min(int *A, int *B, int dim, int *result){

    __shared__ int row_min;
    row_min = *result;
    __syncthreads();

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < dim){
        atomicMin(&row_min, A[id]);
        atomicMin(&row_min, B[id]);
    }  

    __syncthreads();

    // se for primeira thread do bloco
    if(id == blockDim.x * blockIdx.x){
        atomicMin(result, row_min);
    }


}   

// <<LENGTH, WIDTH>>
__global__ void find_max(int *A, int *B, int dim, int *result){

    __shared__ int row_max;
    row_max = *result;
    __syncthreads();

    int id = blockIdx.x * blockDim.x + threadIdx.x;

    if(id < dim){
        atomicMax(&row_max, A[id]);
        atomicMax(&row_max, B[id]);
    }  

    __syncthreads();

    // se for primeira thread do bloco
    if(id == blockDim.x * blockIdx.x){
        atomicMax(result, row_max);
    }


}   

int main(int argc, char **argv) {

    // Declara a dimensão da matriz
    int dim;

    // Declara as matrizes
    int *A,*B, *A_device, *B_device;

    // Cria as streams
    hipStream_t stream1_soma, stream2_min, stream3_max;  
    hipStreamCreate(&stream1_soma);    // Stream que realiza a soma
    hipStreamCreate(&stream2_min);     // Stream que realiza o minímo
    hipStreamCreate(&stream3_max);     // Stream que realiza o máximo

    // Declara o acumulador para o produto escalar global o maximo e o minimo
    int somape = 0,         *somape_device; 
    int minimo = 10000,     *minimo_device;
    int maximo = -10000,    *maximo_device;

    // Aloca os resultados no device
    hipMalloc((void**)&somape_device, sizeof(int)); 
    hipMalloc((void**)&minimo_device, sizeof(int));
    hipMalloc((void**)&maximo_device, sizeof(int));

    // Copia para o device
    hipMemcpyAsync(somape_device, &somape, sizeof(int), hipMemcpyHostToDevice, stream1_soma);
    hipMemcpyAsync(minimo_device, &minimo, sizeof(int), hipMemcpyHostToDevice, stream2_min);
    hipMemcpyAsync(maximo_device, &maximo, sizeof(int), hipMemcpyHostToDevice, stream3_max);
    
    // Lê a dimensão das matrizes
    scanf(" %d", &dim);
    

    // Número de elementos da matriz.
    int tam = dim * dim;

    // Aloca as matrizes no host 
    hipHostMalloc((void**)&A, tam * (sizeof(int)), hipHostMallocDefault); 
    hipHostMalloc((void**)&B, tam * (sizeof(int)), hipHostMallocDefault);

    // Aloca as matrizes no device para cada stream
    hipMalloc((void**)&A_device, tam * (sizeof(int))); 
    hipMalloc((void**)&B_device, tam * (sizeof(int)));

    // Lê a matriz A
    for(int i = 0; i < dim; i++) {
        for(int j = 0; j < dim; j++) {
            scanf(" %d",&(A[i*dim+j]));
        }
    }

    // Lê a matriz B
    for(int i = 0; i < dim; i++) {
        for(int j = 0; j < dim; j++) {
            scanf(" %d",&(B[i*dim+j])); 
        }
    }
    
    // Copia a memória para cada stream do device
    hipMemcpyAsync(A_device, A, tam * (sizeof(int)), hipMemcpyHostToDevice);
    hipMemcpyAsync(B_device, B, tam * (sizeof(int)), hipMemcpyHostToDevice);

    sum_prod_scalar <<<dim, dim, 0, stream1_soma>>> (A_device, B_device, tam, somape_device);
    find_min <<<dim, dim, 0, stream2_min>>> (A_device, B_device, tam, minimo_device);
    find_max <<<dim, dim, 0, stream3_max>>> (A_device, B_device, tam, maximo_device);

    hipMemcpyAsync(&somape, somape_device, sizeof(int), hipMemcpyDeviceToHost, stream1_soma);
    hipMemcpyAsync(&minimo, minimo_device, sizeof(int), hipMemcpyDeviceToHost, stream2_min);
    hipMemcpyAsync(&maximo, maximo_device, sizeof(int), hipMemcpyDeviceToHost, stream3_max);
    
    // Sincroniza as streams
    hipStreamSynchronize(stream1_soma);
    hipStreamSynchronize(stream2_min);
    hipStreamSynchronize(stream3_max);

    // Imprime o resultado
    printf("%d %d %d\n",somape, minimo, maximo); 
    
    // Desaloca as matrizes
    hipHostFree(A);
    hipHostFree(B);
    hipFree(A_device);
    hipFree(B_device);
    hipFree(somape_device);
    hipFree(minimo_device);
    hipFree(maximo_device);
    
    // Destroi as streams
    hipStreamDestroy(stream1_soma);
    hipStreamDestroy(stream2_min);
    hipStreamDestroy(stream3_max);

}
